#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <math.h> 
#include <iostream>     // std::cout
#include <algorithm>    // std::sort
#include <vector>       // std::vector
#include <time.h> 
using namespace std;

#define PINNED 0
#define THREADS 1000

struct Point 
{ 
    int val;     // Group of point 
    double x, y;     // Co-ordinate of point 
    double distance; // Distance from test point 
}; 
  
// Used to sort an array of points by increasing 
// order of distance 
bool comparison(Point a, Point b) 
{ 
    return (a.distance < b.distance); 
} 

/**
 * @param arr    refence points
 * @param n      number of reference points
 * @param k      number of points we want to use for the prediction
 * @param p      point we want to predict
 */
int classifyAPoint(Point arr[], int n, int k, Point p) 
{ 
    // Fill distances of all points from p 
    for (int i = 0; i < n; i++) 
        arr[i].distance = 
            sqrt((arr[i].x - p.x) * (arr[i].x - p.x) + 
                 (arr[i].y - p.y) * (arr[i].y - p.y)); 
  
    // Sort the Points by distance from p 
    sort(arr, arr+n, comparison); 
  
    // Now consider the first k elements and only 
    // two groups 
    int freq1 = 0;     // Frequency of group 0 
    int freq2 = 0;     // Frequency of group 1 
    for (int i = 0; i < k; i++) 
    { 
        if (arr[i].val == 0) 
            freq1++; 
        else if (arr[i].val == 1) 
            freq2++; 
    } 
  
    return (freq1 > freq2 ? 0 : 1); 
}

void InitHostInput(Point arr[], int n, Point p, float *ref_points_host_x, float *ref_points_host_y, float *result_prediction_host) {

    for (int i=0; i<n; i++) 
        ref_points_host_x[i] = arr[i].x;
        ref_points_host_y[i] = arr[i].y;
    
}

__global__ void calculateDistance(int n, Point p, float *ref_points_dev_x, float *ref_points_dev_y, float *result_prediction_dev) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Fill distances of all points from p 
    if(i < n) { 
        result_prediction_dev[i] = 
            sqrt((ref_points_dev_x[i] - p.x) * (ref_points_dev_x[i] - p.x) + 
                 (ref_points_dev_y[i] - p.y) * (ref_points_dev_y[i] - p.y)); 
    }
    
}

int classifyAPointCUDA(Point arr[], int n, int k, Point p) 
{ 
    unsigned int N;
    unsigned int numBytes;
    unsigned int nBlocks, nThreads;
    
    float TiempoTotal;
    hipEvent_t E0, E3;
    
    float * ref_points_dev_x   = NULL;
    float * ref_points_dev_y   = NULL;
    float * result_prediction_dev  = NULL;
    
    float * ref_points_host_x   = NULL;
    float * ref_points_host_y = NULL;
    float * result_prediction_host  = NULL;
    
    // numero de Threads
    nThreads = THREADS;

    // numero de Blocks en cada dimension 
    nBlocks = (n+nThreads-1)/nThreads; 
    printf("nBlocks = %d \n", nBlocks);
    
    numBytes = nBlocks * nThreads * sizeof(float);
    printf("numBytes = %d \n", numBytes);
    
    hipEventCreate(&E0);
    hipEventCreate(&E3);
    
    // Obtener Memoria en el host
    ref_points_host_x = (float*) malloc(numBytes); 
    ref_points_host_y = (float*) malloc(numBytes); 
    result_prediction_host = (float*) malloc(numBytes);  
    
    InitHostInput(arr[], n, p, ref_points_host_x, ref_points_host_y, result_prediction_host);
    
    // Obtener Memoria en el device
    hipMalloc((float**)&ref_points_dev_x, numBytes); 
    hipMalloc((float**)&ref_points_dev_y, numBytes); 
    hipMalloc((float**)&result_prediction_dev, numBytes); 
    
    hipEventRecord(E0, 0);
    
    // Copiar datos desde el host en el device 
    hipMemcpy(ref_points_dev_x, ref_points_host_x, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(ref_points_dev_y, ref_points_host_y, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(result_prediction_dev, result_prediction_host,numBytes, hipMemcpyHostToDevice);
    
    // Ejecutar el kernel 
    calculateDistance<<<nBlocks, nThreads>>>(n, p, ref_points_dev_x, ref_points_dev_y, result_prediction_dev);
    
    // Obtener el resultado desde el host 
    hipMemcpy(result_prediction_host, result_prediction_dev, numBytes, hipMemcpyDeviceToHost);
    
    hipEventRecord(E3, 0); hipEventSynchronize(E3);
    
    // Liberar Memoria del device 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipEventElapsedTime(&TiempoTotal,  E0, E3);
    printf("Invocació Kernel <<<nBlocks, nKernels>>> (N): <<<%d, %d>>> (%d)\n", nBlocks, nThreads, n);
    
    printf("Tiempo Global (00): %4.6f milseg\n", TiempoTotal);
        
    free(ref_points_host_x); free(ref_points_host_y); free(result_prediction_host);
      
    return 0;
    
}

void InitKDefecte(int *k) {
    
    // Parameter to decide group of the testing point 
    (*k) = 3;
    
}

void InitTestPointDefecte(struct Point *p) {
    
    //Test Point
    p->x = 2.5; 
    p->y = 7; 
    
}

void InitDefecte(int *k, struct Point *p) {
    InitKDefecte(k);
    InitTestPointDefecte(p);
}

int main(int argc, char** argv)
{
    
    //Es declaren les variables
    int n, k; 
    struct Point p;
    
    //S'inicialitza la K, i les coordenades del Testing point
    if (argc == 1)      { InitDefecte(&k, &p); }
    else if (argc == 2) { k = atoi(argv[1]); InitTestPointDefecte(&p); }
    else if (argc == 4) { k = atoi(argv[1]); p.x = atof(argv[2]); p.y = atof(argv[3]);}
    else { printf("Usage: ./exe k TestPointCoordenadaX TestPointCoordenadaY\n"); exit(0); }
    
    //Es crea l'estructura sobre la qual es vol fer la predicció
    n = 1000000; // Number of data points 
    Point arr[n];
    
    for(int i = 0; i < n; ++i) {
        arr[i].x = rand() % 100; 
        arr[i].y = rand() % 100; 
        arr[i].val = rand() % 2;
    }
    
    printf("k = %d \n", k);
    
    printf("The Testing Point values are:");
    printf(" x = %lf", p.x);
    printf(" and");
    printf(" y = %lf", p.y);
    printf("\n");
    
    // Calculate the time taken by the sequential code: classifyAPoint function 
    clock_t t; 
    t = clock(); 
    int result = classifyAPoint(arr, n, k, p); 
    t = clock() - t; 
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
    
    printf ("The value classified to unknown point"
            " is %d.\n", result);
            
    printf ("Temps seqüencial:"
            " is %lf.\n", time_taken);
    
    printf("---------------------------------------------------------------------- \n");
    
    // Calculate the time taken by the sequential code: classifyAPoint function 
    clock_t t2; 
    t2 = clock();
    int result2 = classifyAPointCUDA(arr, n, k, p);
    t2 = clock() - t2; 
    double time_taken2 = ((double)t2)/CLOCKS_PER_SEC; // in seconds 
    
    printf ("The value classified to unknown point"
            " is %d.\n", result2);
            
    printf ("Temps CUDA:"
            " is %lf.\n", time_taken2);
    
}


